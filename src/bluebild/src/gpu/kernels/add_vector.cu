#include "hip/hip_runtime.h"
#include <algorithm>

#include "bluebild//config.h"
#include "gpu/kernels/add_vector.hpp"
#include "gpu/util/gpu_runtime.hpp"
#include "gpu/util/gpu_runtime_api.hpp"

namespace bluebild {
namespace gpu {

template <typename T>
__global__ static void
add_vector_real_kernel(std::size_t n, const gpu::ComplexType<T> *__restrict__ a,
                       T *b) {
  for (std::size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
       i += gridDim.x * blockDim.x) {
    b[i] += a[i].x;
  }
}

template <typename T>
auto add_vector_real(gpu::StreamType stream, std::size_t n,
                     const gpu::ComplexType<T> *a, T *b) -> void {
  constexpr std::size_t blockSize = 256;
  constexpr std::size_t maxBlocks = 65535;

  dim3 block(blockSize, 1, 1);
  dim3 grid(std::min<std::size_t>(maxBlocks, (n + block.x - 1) / block.x), 1,
            1);
  gpu::launch_kernel(add_vector_real_kernel<T>, grid, block, 0, stream, n, a,
                     b);
}

template auto add_vector_real<float>(gpu::StreamType stream, std::size_t n,
                                     const gpu::ComplexType<float> *a, float *b)
    -> void;

template auto add_vector_real<double>(gpu::StreamType stream, std::size_t n,
                                      const gpu::ComplexType<double> *a,
                                      double *b) -> void;

} // namespace gpu
} // namespace bluebild
