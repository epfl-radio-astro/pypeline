#include "hip/hip_runtime.h"
#include <algorithm>

#include "bluebild/config.h"
#include "gpu/kernels/gemmexp.hpp"
#include "gpu/util/gpu_runtime.hpp"
#include "gpu/util/gpu_runtime_api.hpp"

#include <hipcub/hipcub.hpp>

namespace bluebild {

static __device__ __forceinline__ void calc_sincos(float x, float *sptr,
                                                   float *cptr) {
  sincosf(x, sptr, cptr);
}

static __device__ __forceinline__ void calc_sincos(double x, double *sptr,
                                                   double *cptr) {
  sincos(x, sptr, cptr);
}

namespace {
template <typename T> struct ComplexOp {
  __device__ __forceinline__ ComplexOp() = default;
  __device__ __forceinline__ ComplexOp(T x_, T y_) : x(x_), y(y_) {}
  __device__ __forceinline__ ComplexOp(const gpu::ComplexType<T> &c)
      : x(c.x), y(c.y) {}

  __device__ __forceinline__ ComplexOp<T>
  operator-(const ComplexOp<T> &other) const {
    return ComplexOp{x - other.x, y - other.y};
  }

  __device__ __forceinline__ ComplexOp<T>
  operator+(const ComplexOp<T> &other) const {
    return ComplexOp{x + other.x, y + other.y};
  }

  __device__ __forceinline__ ComplexOp<T>
  operator*(const ComplexOp<T> &other) const {
    return ComplexOp{x * other.x - y * other.y, x * other.y + other.x * y};
  }

  T x, y;
};
} // namespace

template <typename T, size_t BLOCK_THREADS, hipcub::BlockReduceAlgorithm ALGORITHM>
static __global__ void
gemmexp_kernel(size_t nEig, size_t nPixel, size_t nAntenna, T alpha,
               const gpu::ComplexType<T> *__restrict__ vUnbeam, size_t ldv,
               const T *__restrict__ xyz, size_t ldxyz,
               const T *__restrict__ pixelX, const T *__restrict__ pixelY,
               const T *__restrict__ pixelZ, T *__restrict__ out,
               size_t ldout) {
  using BlockReduceType =
      hipcub::BlockReduce<ComplexOp<T>, BLOCK_THREADS, ALGORITHM>;
  __shared__ typename BlockReduceType::TempStorage tmpStorage;

  for (size_t idxEig = blockIdx.y; idxEig < nEig; idxEig += gridDim.y) {
    for (size_t idxPix = blockIdx.x; idxPix < nPixel; idxPix += gridDim.x) {
      const auto pX = pixelX[idxPix];
      const auto pY = pixelY[idxPix];
      const auto pZ = pixelZ[idxPix];

      ComplexOp<T> localSum{0, 0};
      for (size_t idxAnt = threadIdx.x; idxAnt < nAntenna;
           idxAnt += blockDim.x) {
        const auto imag = alpha * (pX * xyz[idxAnt] + pY * xyz[idxAnt + ldxyz] +
                                   pZ * xyz[idxAnt + 2 * ldxyz]);
        ComplexOp<T> sc;
        calc_sincos(imag, &(sc.y), &(sc.x));
        localSum = localSum + sc * vUnbeam[idxEig * ldv + idxAnt];
      }

      auto totalSum = BlockReduceType(tmpStorage).Sum(localSum);
      if (threadIdx.x == 0) {
        out[idxEig * ldout + idxPix] =
            totalSum.x * totalSum.x + totalSum.y * totalSum.y;
      }
    }
  }
}

template <typename T>
auto gemmexp_gpu(gpu::StreamType stream, std::size_t nEig, std::size_t nPixel,
                 std::size_t nAntenna, T alpha,
                 const gpu::ComplexType<T> *vUnbeam, std::size_t ldv,
                 const T *xyz, std::size_t ldxyz, const T *pixelX,
                 const T *pixelY, const T *pixelZ, T *out, std::size_t ldout)
    -> void {
  constexpr std::size_t blockSize = 512;
  constexpr std::size_t maxBlocks = 65535;

  dim3 block(blockSize, 1, 1);
  dim3 grid(std::min<std::size_t>(maxBlocks, nPixel),
            std::min<std::size_t>(maxBlocks, nEig), 1);

  gpu::launch_kernel(
      gemmexp_kernel<T, blockSize,
                     hipcub::BlockReduceAlgorithm::BLOCK_REDUCE_WARP_REDUCTIONS>,
      grid, block, 0, stream, nEig, nPixel, nAntenna, alpha, vUnbeam, ldv, xyz,
      ldxyz, pixelX, pixelY, pixelZ, out, ldout);
}

template auto
gemmexp_gpu<float>(gpu::StreamType stream, std::size_t nEig, std::size_t nPixel,
                   std::size_t nAntenna, float alpha,
                   const gpu::ComplexType<float> *__restrict__ vUnbeam,
                   std::size_t ldv, const float *__restrict__ xyz,
                   std::size_t ldxyz, const float *__restrict__ pixelX,
                   const float *__restrict__ pixelY,
                   const float *__restrict__ pixelZ, float *__restrict__ out,
                   std::size_t ldout) -> void;

template auto
gemmexp_gpu<double>(gpu::StreamType stream, std::size_t nEig,
                    std::size_t nPixel, std::size_t nAntenna, double alpha,
                    const gpu::ComplexType<double> *__restrict__ vUnbeam,
                    std::size_t ldv, const double *__restrict__ xyz,
                    std::size_t ldxyz, const double *__restrict__ pixelX,
                    const double *__restrict__ pixelY,
                    const double *__restrict__ pixelZ, double *__restrict__ out,
                    std::size_t ldout) -> void;
} // namespace bluebild
